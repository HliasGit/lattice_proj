#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vec_min(int *a, int *b, int size) {
    extern __shared__ int sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sdata[tid] = (i < size) ? a[i] : INT_MAX;
    __syncthreads();

    // Block reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s>>=1) {
        if (tid < s) {
            sdata[tid] = (sdata[tid] < sdata[tid + s]) ? sdata[tid] : sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        b[blockIdx.x] = sdata[0];
    }
}


int main(){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    // sizes
    int vecSize = 1024*1024;
    int n_threads = 1024;
    int n_blocks = vecSize/n_threads;

    // Allocate memory for GPU
    int *A;
    int *B;

    // Allocate for host
    int data[vecSize];
    int result[n_threads];

    // Initialize data
    for(int i=0; i<vecSize; i++){
        data[i] = i+4;
    }

    // Set a minimum somewhere
    data[124737] = 1;

    // Allocate CUDA memory
    hipMalloc(&A, vecSize*sizeof(int));
    hipMalloc(&B, n_threads*sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy to the CUDA mem
    hipMemcpy(A, data, vecSize*sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start);
    vec_min<<<n_blocks, n_threads, n_threads * sizeof(int)>>>(A, B, vecSize);
    hipDeviceSynchronize(),
    hipEventRecord(stop);

    // Copy back
    hipMemcpy(result, B, n_blocks*sizeof(int), hipMemcpyDeviceToHost);

    int min = INT_MAX;
    for (int i = 0; i < n_blocks; i++) {
        if (result[i] < min) {
            min = result[i];
        }
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Minimum value: " << min << std::endl;
    std::cout << "Time: " << milliseconds / 1000<< "s" << std::endl;

    hipFree(A);
    hipFree(B);   

}